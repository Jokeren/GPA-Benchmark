#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_pipeline.h>

__global__
void tensor_transpose(int dim_input, int dim_output, int nblocks, int tile_size,
  double *input, double *output) {
  extern __shared__ double tile[];
  int block_idx = blockIdx.x;
  int phase = 0;

  for (int i = threadIdx.x; i < tile_size; i += blockDim.x) {
    __pipeline_memcpy_async(&tile[phase * TILE_SIZE + i], &input[i + block_idx * tile_size], sizeof(double));
  }
  __pipeline_commit();

  for (; block_idx < nblocks; block_idx += gridDim.x) {
    int it = block_idx, im = 0, offset1 = 0;

    if (block_idx + gridDim.x < nblocks) {
      int p = 1 - phase;
      for (int i = threadIdx.x; i < tile_size; i += blockDim.x) {
        __pipeline_memcpy_async(&tile[p * TILE_SIZE + i], &input[i + (block_idx + gridDim.x) * tile_size], sizeof(double));
      }
      __pipeline_commit();
    }
    
    for (int i = 0; i < dim_input; i++) {
      im = it * d_shape_input_r[i];
      offset1 += d_stride_input[i] * (it - im * d_shape_input[i]);
      it = im;
    }

    if (block_idx + gridDim.x < nblocks) {
      __pipeline_wait_prior(1);
    } else {
      __pipeline_wait_prior(0);
    }
    __syncthreads();
  
    for (int i = threadIdx.x; i < tile_size; i += blockDim.x) {
      it = i;
      int offset2 = 0, local_offset = 0;
      for (int j = 0; j < dim_output; j++) {
        im = it * d_shape_output_r[j];
        int tmp = it - im * d_shape_output[j];
        offset2 += d_stride_output_global[j] * tmp;
        local_offset += d_stride_output_local[j] * tmp;
        it = im;
      }
      output[offset1 + offset2] = tile[phase * TILE_SIZE + local_offset];
    }

    phase = 1 - phase;
    __syncthreads();
  }
}
