#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_pipeline.h>

using namespace nvcuda::experimental;


#define TILE_SIZE 5900
#define NTHREADS 512

__global__
void tensor_transpose(int dim_input, int dim_output, int nblocks, int tile_size,
  double *input, double *output) {
  pipeline pipe;
  extern __shared__ double tile[];
  int block_idx = blockIdx.x;
  int iters = 0;
  int phase = 0;

  for (int i = threadIdx.x; i < tile_size; i += blockDim.x) {
    memcpy_async(tile[phase * TILE_SIZE + i], input[i + block_idx * tile_size], pipe);
  }
  pipe.commit();

  for (; block_idx < nblocks; block_idx += gridDim.x) {
    ++iters;
    int it = block_idx, im = 0, offset1 = 0;

    if (block_idx + gridDim.x < nblocks) {
      int p = 1 - phase;
      for (int i = threadIdx.x; i < tile_size; i += blockDim.x) {
        memcpy_async(tile[p * TILE_SIZE + i], input[i + (block_idx + gridDim.x) * tile_size], pipe);
      }
      pipe.commit();
    }
    
    for (int i = 0; i < dim_input; i++) {
      im = it * d_shape_input_r[i];
      offset1 += d_stride_input[i] * (it - im * d_shape_input[i]);
      it = im;
    }

    pipe.wait(iters - 1);
    __syncthreads();
  
    for (int i = threadIdx.x; i < tile_size; i += blockDim.x) {
      it = i;
      int offset2 = 0, local_offset = 0;
      for (int j = 0; j < dim_output; j++) {
        im = it * d_shape_output_r[j];
        int tmp = it - im * d_shape_output[j];
        offset2 += d_stride_output_global[j] * tmp;
        local_offset += d_stride_output_local[j] * tmp;
        it = im;
      }
      output[offset1 + offset2] = tile[phase * TILE_SIZE + local_offset];
    }

    phase = 1 - phase;
    __syncthreads();
  }
}
