#include "hip/hip_runtime.h"
/*********************************************************************************
Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
All rights reserved.
  
Permission to use, copy, modify and distribute this software and its documentation for 
educational purpose is hereby granted without fee, provided that the above copyright 
notice and this permission notice appear in all copies of this software and that you do 
not sell the software.
  
THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
OTHERWISE.

The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
**********************************************************************************/
#ifndef _KERNEL_H_
#define _KERNEL_H_

__global__ void
Kernel( Node* g_graph_nodes, int* g_graph_edges, bool* g_graph_mask, bool* g_updating_graph_mask, bool *g_graph_visited, int* g_cost, int no_of_nodes) 
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
  if( tid<no_of_nodes && g_graph_mask[tid])
  {
    g_graph_mask[tid]=false;
    int nedges = g_graph_nodes[tid].no_of_edges;
    int left = nedges - nedges / 4 * 4;
    int cost = g_cost[tid];
    for (int i=g_graph_nodes[tid].starting; i<(left + g_graph_nodes[tid].starting); i+=1)
    {   
      int id1 = g_graph_edges[i];
      if(!g_graph_visited[id1])
      {   
        g_cost[id1]=cost + 1;
        g_updating_graph_mask[id1]=true;
      }   
    }   
    for (int i=g_graph_nodes[tid].starting + left; i<(nedges + g_graph_nodes[tid].starting); i+=4)
    {   
      int id1 = g_graph_edges[i];
      int id2 = g_graph_edges[i + 1]; 
      int id3 = g_graph_edges[i + 2]; 
      int id4 = g_graph_edges[i + 3]; 
      if(!g_graph_visited[id1])
      {   
        g_cost[id1]= cost + 1;
        g_updating_graph_mask[id1]=true;
      }   
      if (!g_graph_visited[id2]) {
        g_cost[id2]= cost + 1;
        g_updating_graph_mask[id2]=true;
      }   
      if (!g_graph_visited[id3]) {
        g_cost[id3]= cost + 1;
        g_updating_graph_mask[id3]=true;
      }   
      if (!g_graph_visited[id4]) {
        g_cost[id4]= cost + 1;
        g_updating_graph_mask[id4]=true;
      }   
    }   
  }
}

#endif 
