#include "hip/hip_runtime.h"
//========================================================================================================================================================================================================200
//	findRangeK function
//========================================================================================================================================================================================================200

__global__ void 
findRangeK(	long height,

			knode *knodesD,
			long knodes_elem,

			long *currKnodeD,
			long *offsetD,
			long *lastKnodeD,
			long *offset_2D,
			int *startD,
			int *endD,
			int *RecstartD, 
			int *ReclenD)
{

	// private thread IDs
	int thid = threadIdx.x;
	int bid = blockIdx.x;
  int start = startD[bid];
  int end = endD[bid];
  int *curr_keys = knodesD[currKnodeD[bid]].keys;
  int *last_keys = knodesD[lastKnodeD[bid]].keys;
  int *curr_indices = knodesD[currKnodeD[bid]].indices;
  int *last_indices = knodesD[lastKnodeD[bid]].indices;

	// ???
	int i;
	for(i = 0; i < height; i++){
		if((curr_keys[thid] <= start) && (curr_keys[thid+1] > start)){
			// this conditional statement is inserted to avoid crush due to but in original code
			// "offset[bid]" calculated below that later addresses part of knodes goes outside of its bounds cause segmentation fault
			// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
			if(curr_indices[thid] < knodes_elem){
				offsetD[bid] = curr_indices[thid];
			}
		}
		if((last_keys[thid] <= end) && (last_keys[thid+1] > end)){
			// this conditional statement is inserted to avoid crush due to but in original code
			// "offset_2[bid]" calculated below that later addresses part of knodes goes outside of its bounds cause segmentation fault
			// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
			if(last_indices[thid] < knodes_elem){
				offset_2D[bid] = last_indices[thid];
			}
		}
		__syncthreads();

		// set for next tree level
    curr_keys = knodesD[offsetD[bid]].keys;
    curr_indices = knodesD[offsetD[bid]].indices;
    last_keys = knodesD[offset_2D[bid]].keys;
    last_indices = knodesD[offset_2D[bid]].indices;

		__syncthreads();
	}

  int kend = last_keys[thid];
	// Find the index of the starting record
	if(curr_keys[thid] == start){
		RecstartD[bid] = curr_indices[thid];
	}
	__syncthreads();

	// Find the index of the ending record
	if(kend == end){
		ReclenD[bid] = last_indices[thid] - RecstartD[bid]+1;
	}

}

//========================================================================================================================================================================================================200
//	End
//========================================================================================================================================================================================================200
